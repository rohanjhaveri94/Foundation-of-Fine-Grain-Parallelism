#include "hip/hip_runtime.h"
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* gemm.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "gemm.h"
double cpuSecond()
{
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return((double)tp.tv_sec + (double)tp.tv_usec*1e-6);
}

/* Array initialization. */
static
void init_array(int ni, int nj, int nk,
		DATA_TYPE *alpha,
		DATA_TYPE *beta,
		DATA_TYPE POLYBENCH_2D(C,NI,NJ,ni,nj),
		DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nk),
		DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj))
{
  int i, j;

  *alpha = 1.5;
  *beta = 1.2;
  for (i = 0; i < ni; i++)
    for (j = 0; j < nj; j++)
      C[i][j] = (DATA_TYPE) ((i*j+1) % ni) / ni;
  for (i = 0; i < ni; i++)
    for (j = 0; j < nk; j++)
      A[i][j] = (DATA_TYPE) (i*(j+1) % nk) / nk;
  for (i = 0; i < nk; i++)
    for (j = 0; j < nj; j++)
      B[i][j] = (DATA_TYPE) (i*(j+2) % nj) / nj;
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int ni, int nj,
		 DATA_TYPE POLYBENCH_2D(C,NI,NJ,ni,nj))
{
  int i, j;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("C");
  for (i = 0; i < ni; i++)
    for (j = 0; j < nj; j++) {
	if ((i * ni + j) % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
	fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, C[i][j]);
    }
  POLYBENCH_DUMP_END("C");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_gemm(int ni, int nj, int nk,
		 DATA_TYPE alpha,
		 DATA_TYPE beta,
		 DATA_TYPE POLYBENCH_2D(C,NI,NJ,ni,nj),
		 DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nk),
		 DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj))
{
 // int i, j, k;

//BLAS PARAMS
//TRANSA = 'N'
//TRANSB = 'N'
// => Form C := alpha*A*B + beta*C,
//A is NIxNK
//B is NKxNJ
//C is NIxNJ

__global__ void matmul (int **a, int **b, int **c, int _PB_NI, int _PB_NJ, int _PB_NK, DATA_TYPE alpha) ;

#ifdef ceild
# undef ceild
#endif
#ifdef floord
# undef floord
#endif
#ifdef max
# undef max
#endif
#ifdef min
# undef min
#endif
#define ceild(x,y) (((x) > 0)? (1 + ((x) - 1)/(y)): ((x) / (y)))
#define floord(x,y) (((x) > 0)? ((x)/(y)): 1 + (((x) -1)/ (y)))
#define max(x,y)    ((x) > (y)? (x) : (y))
#define min(x,y)    ((x) < (y)? (x) : (y))
/* Copyright (C) 1991-2016 Free Software Foundation, Inc.
   This file is part of the GNU C Library.

   The GNU C Library is free software; you can redistribute it and/or
   modify it under the terms of the GNU Lesser General Public
   License as published by the Free Software Foundation; either
   version 2.1 of the License, or (at your option) any later version.

   The GNU C Library is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
   Lesser General Public License for more details.

   You should have received a copy of the GNU Lesser General Public
   License along with the GNU C Library; if not, see
   <http://www.gnu.org/licenses/>.  */
/* This header is separate from features.h so that the compiler can
   include it implicitly at the start of every compilation.  It must
   not itself include <features.h> or any other header that includes
   <features.h> because the implicit include comes before any feature
   test macros that may be defined in a source file before it first
   explicitly includes a system header.  GCC knows the name of this
   header in order to preinclude it.  */
/* glibc's intent is to support the IEC 559 math functionality, real
   and complex.  If the GCC (4.9 and later) predefined macros
   specifying compiler intent are available, use them to determine
   whether the overall intent is to support these features; otherwise,
   presume an older compiler has intent to support these features and
   define these macros by default.  */
/* wchar_t uses Unicode 9.0.0.  Version 9.0 of the Unicode Standard is
   synchronized with ISO/IEC 10646:2014, fourth edition, plus
   Amd. 1  and Amd. 2 and 273 characters from forthcoming  10646, fifth edition.
   (Amd. 2 was published 2016-05-01,
   see https://www.iso.org/obp/ui/#iso:std:iso-iec:10646:ed-4:v1:amd:2:v1:en) */
/* We do not support C11 <threads.h>.  */
//  register int lbv, ubv, lb, ub, lb1, ub1, lb2, ub2;
//  register int c0, c1, c2, c3, c4, c5;
  
	int **d_a, **d_b, **d_c; 
	double iStart, iElaps;
	_PB_NI = 1024;
	_PB_NJ = 1024;
	_PB_NK = 1024;
	
	// initialize matrices a and b with appropriate values 
	int size_A = _PB_NI * _PB_NI * sizeof(int); 
	int size_B = _PB_NJ * _PB_NJ * sizeof(int); 
	int size_C = _PB_NK * _PB_NK * sizeof(int); 
	
	hipMalloc((void **) &d_a, size_A); 
	hipMalloc((void **) &d_b, size_B); 
	hipMalloc((void **) &d_c, size_C); 
	
	hipMemcpy(d_a, A, size_A, hipMemcpyHostToDevice); 
	hipMemcpy(d_b, B, size_B, hipMemcpyHostToDevice); 
	
	dim3 dimGrid(60, 1); 
	dim3 dimBlock(1024, 1024); 
	iStart = cpuSecond();
	matmul<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, _PB_NI,_PB_NJ,_PB_NK,alpha); 
	hipDeviceSynchronize();
	iElaps = cpuSecond() - iStart;
	printf("Time elapsed %f sec\n", iElaps);

	hipMemcpy(C, d_c, size_C, hipMemcpyDeviceToHost); 
	hipFree(d_a); 
	hipFree(d_b); 
	hipFree(d_c); 
	
  
  

}

__global__ void matmul (int **A, int **B, int **C, int _PB_NI, int _PB_NJ, int _PB_NK, DATA_TYPE alpha) 
{  
	
	int c2, c3, c4, c5 = 0;
	
	int col = threadIdx.x + blockDim.x * blockIdx.x; 
	int row = threadIdx.y + blockDim.y * blockIdx.y; 
	
	if (col < _PB_NI && row < _PB_NJ){
		
		for(c2 = 0; c2 <= floord((_PB_NK -1), 4); c2++ ){
		
			for (c3 = (4 * row); c3 <= min((_PB_NI + -1), ((4 * row) + 3)); c3++)
			{
			  for (c4 = (4 * col); c4 <= min((_PB_NJ + -1), ((4 * col) + 3)); c4++)
			  {
					for (c5 = (4 * c2); c5 <= min((_PB_NK + -1), ((4 * c2) + 3)); c5++) {
					
						C[c3][c4] += alpha * A[c3][c5] * B[c5][c4];
					}
				}
			}
			
		}
	}
}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int ni = NI;
  int nj = NJ;
  int nk = NK;

  /* Variable declaration/allocation. */
  DATA_TYPE alpha;
  DATA_TYPE beta;
  POLYBENCH_2D_ARRAY_DECL(C,DATA_TYPE,NI,NJ,ni,nj);
  POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,NI,NK,ni,nk);
  POLYBENCH_2D_ARRAY_DECL(B,DATA_TYPE,NK,NJ,nk,nj);

  /* Initialize array(s). */
  init_array (ni, nj, nk, &alpha, &beta,
	      POLYBENCH_ARRAY(C),
	      POLYBENCH_ARRAY(A),
	      POLYBENCH_ARRAY(B));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_gemm (ni, nj, nk,
	       alpha, beta,
	       POLYBENCH_ARRAY(C),
	       POLYBENCH_ARRAY(A),
	       POLYBENCH_ARRAY(B));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(ni, nj,  POLYBENCH_ARRAY(C)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(C);
  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(B);

  return 0;
}
